#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <locale.h>
#include <iostream>

int n = 3; // ����������� ������������
double a[] = { 0, 0, 0 }; // ������ ������� �������� ����������
double b[] = { 100, 100, 100 }; // ������� ������� �������� ����������
int m[] = { 10, 10, 10 }; // ���������� ��������� �������
double e1 = 1e-7; // �������� ���������� 
double e2 = 1e-7; // �������� ���������� 
double t = 0.5; // �������� ����

double p[] = { 10, 20, 30 };

// ����������� �������
__device__ double f(double *x, double *p, int n, int id)
{
	double s = 0.0;
	for (auto i = 0; i < n; i++) s += (x[i + n*id] - p[i])*(x[i + n*id] - p[i]);
	return s;
}

// �������� ����������� �������
__device__ void gradientvector(double *g, double *x, double *p, int n, int id)
{
	for (auto i = 0; i < n; i++)
	{
		g[i + n*id] = 2.0*(x[i + n*id] - p[i]);
	}
}

__device__ double l2(double *x, double *x1, int n, int id)
{
	double s = 0;
	for (auto i = 0; i < n; i++) s += (x[i + n*id] - x1[i + n*id])*(x[i + n*id] - x1[i + n*id]);
	return s;
}

// ������������� ���������� ��������������� �����
__global__ void setuprand(hiprandState *state, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id<total; id += gridDim.x*blockDim.x)
		hiprand_init(1234, id, 0, &state[id]);
}

__global__ void fillindex(int *k, int *m, int n, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id<total; id += gridDim.x*blockDim.x)
	{
		for (int i = 0, j = id; i < n; i++)
		{
			k[n*id + i] = j%m[i];
			j /= m[i];
		}
	}
}

__global__ void cyclicalgorithm(double *fx, double *x, double *g, double *fx1, double *x1, double *x2, double *a, double *b, double *p, int *k, int *m, int n, double t, double e1, double e2, hiprandState *state, int total)
{
	// ����� ��������� �����
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < total; id += gridDim.x*blockDim.x)
	{
		hiprandState localState = state[id];
		for (int i = 0; i < n; i++)
		{
			double p = hiprand_uniform_double(&localState);
			x[n*id + i] = ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i] + p*(b[i] - a[i])) / m[i];
		}
		state[id] = localState;
	}
	// ���������� �������
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < total; id += gridDim.x*blockDim.x)
	{
		fx[id] = f(x, p, n, id);
	}

	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < total; id += gridDim.x*blockDim.x)
	{
		for (;;)
		{
			for (int j = 0; j < n; j++) x2[n*id + j] = x[n*id + j];
				
			// ���� �� ����������
			for (int i = 0; i < n; i++)
			{
				// �������� ���������� ����������� �� �����������

				// ���������� ���������
				gradientvector(g, x, p, n, id);

				double s = (g[n*id + i]>-g[n*id + i]) ? g[n*id + i] : -g[n*id + i];
				if (s < e1) continue;
				for (double t1 = t;; t1 /= 2)
				{
					for (int j = 0; j < n; j++) x1[n*id + j] = x[n*id + j];
					x1[n*id + i] = x[n*id + i] - t1*g[n*id + i];
					if (x1[n*id + i] < ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i]) / m[i])
						x1[n*id + i] = ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i]) / m[i];
					if (x1[n*id + i] > ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i] + b[i] - a[i]) / m[i])
						x1[n*id + i] = ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i] + b[i] - a[i]) / m[i];
					fx1[id] = f(x1, p, n, id);
					if (fx1[id] < fx[id])
					{
						x[n*id + i] = x1[n*id + i];
						fx[id] = fx1[id];
						break;
					}
					s = (x[n*id + i] > x1[n*id + i]) ? (x[n*id + i] - x1[n*id + i]) : (x1[n*id + i] - x[n*id + i]);
					if (s < e2) break;
				}
			}
			if (l2(x, x2, n, id) < e2*e2) break;
		}
	}

}
int main()
{
	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	int total = 1; for (auto i = 0; i < n; i++) total *= m[i];
	int N = (1 + sqrt(total)>255) ? 255 : (int)(1 + sqrt(total));

	hiprandState *devStates;
	hipMalloc((void **)&devStates, total*sizeof(hiprandState));
	setuprand <<<1, N>>>(devStates, total);

	double *devA, *devB, *devG, *devX, *devX1, *devX2, *devP, *devFX, *devFX1;
	double *x, *fx;
	int *devK, *devM;
	hipMalloc((void **)&devA, n*sizeof(double));
	hipMalloc((void **)&devB, n*sizeof(double));
	hipMalloc((void **)&devP, n*sizeof(double));
	hipMalloc((void **)&devM, n*sizeof(int));
	hipMalloc((void **)&devK, n*total*sizeof(int));
	hipMalloc((void **)&devG, n*total*sizeof(double));
	hipMalloc((void **)&devX, n*total*sizeof(double));
	hipMalloc((void **)&devX1, n*total*sizeof(double));
	hipMalloc((void **)&devX2, n*total*sizeof(double));
	hipMalloc((void **)&devFX, total*sizeof(double));
	hipMalloc((void **)&devFX1, total*sizeof(double));
	fx = (double *)malloc(total*sizeof(double));
	x = (double *)malloc(n*sizeof(double));

	hipMemcpy(devA, a, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devP, p, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devM, m, n*sizeof(int), hipMemcpyHostToDevice);

	fillindex <<<1, N>>>(devK, devM, n, total);

	cyclicalgorithm <<<1, N>>>(devFX, devX, devG, devFX1, devX1, devX2, devA, devB, devP, devK, devM, n, t, e1, e2, devStates, total);


	hipMemcpy(fx, devFX, total*sizeof(double), hipMemcpyDeviceToHost);
	// ���������� ����������� ��������
	int index = 0;
	for (int id = 1; id < total; id++)
	{
		if (fx[id] < fx[index])
			index = id;
	}
	hipMemcpy(x, &devX[n*index], n*sizeof(double), hipMemcpyDeviceToHost);

	// ����� �����������

	std::cout << "����� �������� : ";
	for (auto i = 0; i < n; i++)
	{
		std::cout << x[i];
		if (i < n - 1) std::cout << ",";
	}
	std::cout << std::endl;

	std::cout << "�������� �������� : " << fx[index] << std::endl;

	free(x);
	free(fx);
	hipFree(devG);
	hipFree(devX);
	hipFree(devX1);
	hipFree(devX2);
	hipFree(devFX);
	hipFree(devFX1);
	hipFree(devA);
	hipFree(devB);
	hipFree(devP);
	hipFree(devM);
	hipFree(devK);
	hipFree(devStates);

	getchar(); // �������� ����� � ���������� ����� ����������� ���������
	return 0;
}

