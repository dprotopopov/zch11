
#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <locale.h>
#include <iostream>

int n = 3; // ����������� ������������
double a[] = { 0, 0, 0 }; // ������ ������� �������� ����������
double b[] = { 100, 100, 100 }; // ������� ������� �������� ����������
int m[] = { 10, 10, 10 }; // ���������� ��������� �������
int R = 10000; // ���������� ��������

double p[] = { 10, 20, 30 };

// ����������� �������
// ����� ������ ���� ������� �������� ������� s
__global__ void f(double *fx, double *x, double *p, int n, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id<total; id += gridDim.x*blockDim.x)
	{
		fx[id] = 0;	
		for (int i = 0; i<n; i++)
		{
			fx[id] += (x[n*id + i] - p[i])*(x[n*id + i] - p[i]);				
		}
	}
}

// ������������� ���������� ��������������� �����
__global__ void setuprand(hiprandState *state, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id<total; id += gridDim.x*blockDim.x)
		hiprand_init(1234, id, 0, &state[id]);
}

__global__ void fillindex(int *k, int *m, int n, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id<total; id += gridDim.x*blockDim.x)
	{
		for (int i = 0, j = id; i < n; i++)
		{
			k[n*id + i] = j%m[i];
			j /= m[i];
		}		
	}
}
// ��������� ���������������� �������
__global__ void randvector(double *x, double *a, double *b, int *k, int *m, int n, hiprandState *state, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < total; id += gridDim.x*blockDim.x)
	{
		hiprandState localState = state[id];
		for (int i = 0; i < n; i++)
		{
			double p = hiprand_uniform_double(&localState);
			x[n*id + i] = ((m[i] - k[n*id + i])*a[i] + k[n*id + i] * b[i] + p*(b[i] - a[i])) / m[i];
		}
		state[id] = localState;
	}
}

__global__ void getminimal(double *fx, double *fx1, double *x, double *x1, int n, int total)
{
	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < total; id += gridDim.x*blockDim.x)
	{
		if (fx1[id] < fx[id])
		{
			fx[id] = fx1[id];
			for (int i = 0; i < n; i++) x[n*id + i] = x1[n*id + i];
		}
	}
}

int main()
{
	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	int total = 1; for (auto i = 0; i < n; i++) total *= m[i];
	int N = (1 + sqrt(total)>255) ? 255 : (int)(1 + sqrt(total));

	hiprandState *devStates;
	hipMalloc((void **)&devStates, total*sizeof(hiprandState));
	setuprand <<<1, N>>>(devStates, total);

	double *devA, *devB, *devX, *devX1, *devP, *devFX, *devFX1;
	double *x, *fx;
	int *devK, *devM;
	hipMalloc((void **)&devA, n*sizeof(double));
	hipMalloc((void **)&devB, n*sizeof(double));
	hipMalloc((void **)&devP, n*sizeof(double));
	hipMalloc((void **)&devM, n*sizeof(int));
	hipMalloc((void **)&devK, n*total*sizeof(int));
	hipMalloc((void **)&devX, n*total*sizeof(double));
	hipMalloc((void **)&devX1, n*total*sizeof(double));
	hipMalloc((void **)&devFX, total*sizeof(double));
	hipMalloc((void **)&devFX1, total*sizeof(double));
	fx = (double *)malloc(total*sizeof(double));
	x = (double *)malloc(n*sizeof(double));

	hipMemcpy(devA, a, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devP, p, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devM, m, n*sizeof(int), hipMemcpyHostToDevice);

	fillindex <<<1, N>>>(devK, devM, n, total);

	// ����� ��������� �����
	randvector <<<1, N>>>(devX, devA, devB, devK, devM, n, devStates, total);
	f <<<1, N>>>(devFX, devX, devP, n, total);

	for (auto r = 0; r < R; r++)
	{
		//std::clog << "����� ��������� �����" << std::endl;
		randvector <<<1, N>>>(devX1, devA, devB, devK, devM, n, devStates, total);
		f <<<1, N>>>(devFX1, devX1, devP, n, total);
		getminimal <<<1, N>>>(devFX, devFX1, devX, devX1, n, total);
	}

	hipMemcpy(fx, devFX, total*sizeof(double), hipMemcpyDeviceToHost);
	// ���������� ����������� ��������
	int index = 0;
	for (int id = 1; id < total; id++)
	{
		if (fx[id] < fx[index])
			index = id;
	}
	hipMemcpy(x, &devX[n*index], n*sizeof(double), hipMemcpyDeviceToHost);

	// ����� �����������

	std::cout << "����� �������� : ";
	for (auto i = 0; i < n; i++)
	{
		std::cout << x[i];
		if (i < n - 1) std::cout << ",";
	}
	std::cout << std::endl;

	std::cout << "�������� �������� : " << fx[index] << std::endl;

	free(x);
	free(fx);
	hipFree(devX);
	hipFree(devX1);
	hipFree(devFX);
	hipFree(devFX1);
	hipFree(devA);
	hipFree(devB);
	hipFree(devP);
	hipFree(devM);
	hipFree(devK);
	hipFree(devStates);

	getchar(); // �������� ����� � ���������� ����� ����������� ���������
	return 0;
}

